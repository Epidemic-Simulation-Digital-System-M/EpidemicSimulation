#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <device_atomic_functions.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include "lib/cJSON.h"
#include <emmintrin.h>
#include <immintrin.h>


#define MAX_NODES 1000
#define MAX_EDGES 10000


int* N; // Indici dell'inizio dei vicini per ogni nodo
int* L; // Lista di adiacenza compressa
int* Levels; // Momento dell'infezione: istante in cui viene infettato
bool* Immune; // Stato di immunità

int num_nodes;
int num_edges;
int size_L;

double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double)ts.tv_sec + (double)ts.tv_nsec * 1.e-9);
}

char* read_file(const char* filename) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        printf("Error opening file!\n");
        return NULL;
    }

    char* json_string = NULL;
    size_t size = 0;
    size_t capacity = 128;  // Initial buffer size
    json_string = (char*)malloc(capacity);
    if (!json_string) {
        printf("Memory allocation failed!\n");
        fclose(file);
        return NULL;
    }

    int ch;
    while ((ch = fgetc(file)) != EOF) {
        json_string[size++] = (char)ch;
        // Resize buffer if needed
        if (size >= capacity - 1) {
            capacity *= 2;  // Double the buffer size
            json_string = (char*)realloc(json_string, capacity);
            if (!json_string) {
                printf("Memory reallocation failed!\n");
                fclose(file);
                return NULL;
            }
        }
    }
    json_string[size] = '\0';  // Null-terminate the string
    fclose(file);
    return json_string;
}

void import_network(const char* filename) {
    char filepath[256];
    snprintf(filepath, sizeof(filepath), "../../GRAPH_GENERATOR/%s", filename);
    char* json_string = read_file(filepath);
    if (!json_string) {
        exit(1);
    }

    cJSON* root = cJSON_Parse(json_string);
    free(json_string);  // Free memory after parsing
    if (!root) {
        printf("Error parsing JSON!\n");
        return;
    }

    cJSON* json_numNodes = cJSON_GetObjectItem(root, "num_nodes");
    cJSON* json_numEdges = cJSON_GetObjectItem(root, "num_edges");
    num_nodes = json_numNodes->valueint;
    num_edges = json_numEdges->valueint;

    // Extract arrays
    cJSON* json_N = cJSON_GetObjectItem(root, "N");
    cJSON* json_L = cJSON_GetObjectItem(root, "L");

    int size_N = cJSON_GetArraySize(json_N);
    size_L = cJSON_GetArraySize(json_L);

    N = (int*)malloc(size_N * sizeof(int));
    L = (int*)malloc(size_L * sizeof(int));
    Levels = (int*)malloc(num_nodes * sizeof(int));
    Immune = (bool*)malloc(num_nodes * sizeof(bool));

    for (int i = 0; i < size_N; i++) {
        N[i] = cJSON_GetArrayItem(json_N, i)->valueint;
    }
    for (int i = 0; i < size_L; i++) {
        L[i] = cJSON_GetArrayItem(json_L, i)->valueint;
    }

    for (int i = 0;i < num_nodes;i++) {
        Levels[i] = -1; // Non infetto
        Immune[i] = false;  // Non immune
    }
    Levels[0] = 0; // Nodo inizialmente infetto al tempo 0
}

void print_network() {
    printf("Network:\n");
    for (int i = 0; i < num_nodes; i++) {
        printf("%d: ", i);
        for (int j = N[i]; j < N[i + 1]; j++) {
            printf("%d ", L[j]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_status(int step, int active_infections, int* d_Levels) {
    printf("Step %d: %d active infections\n", step, active_infections);
    if (active_infections > 0) {
        hipMemcpy(Levels, d_Levels, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
        printf("Infected nodes: ");
        for (int i = 0; i < num_nodes; i++) {
            if (Levels[i] == step) {
                printf("%d ", i);
            }
        }
        printf("\n");
    }
}


__global__ void simulate_step(int* d_N, int* d_L, int* d_Levels, bool* d_Immune, int num_nodes, double p, double q, int step, int* d_active_infections) {
    int tid_in_warp = (threadIdx.x) % 32;
    
    int warp_id = (threadIdx.x + blockIdx.x*blockDim.x) / 32;

    int start_index = warp_id * 32;

    int final_index = start_index + 32;
    if (final_index > num_nodes) {
        final_index = num_nodes;
    }

    hiprandState state;
    bool is_init = false;

    for (int i = start_index; i < final_index; i++) {
        if (d_Levels[i] == step) { //Il nodo è infetto
            if (!is_init) {
                hiprand_init(0, threadIdx.x, 0, &state);
                is_init = true;
            }
            for (int j = d_N[i] + tid_in_warp; j < d_N[i + 1]; j += 32) {
                int neighbor = d_L[j];
                //printf("Thread %d: Nodo %d Vicino %d\n", tid_in_warp,i, neighbor);
                if (d_Levels[neighbor] == -1 && !d_Immune[neighbor] && (hiprand_uniform(&state) < p)) {
                    // Infetto al prossimo step
                    // Usa atomicCAS per evitare doppie infezioni
                    int old_level = atomicCAS(&d_Levels[neighbor], -1, step + 1);
                    if (old_level == -1) {  // Solo il primo thread che infetta il nodo lo conta
                        atomicAdd(d_active_infections, 1);
                        //printf("Thread %d Blocco %d: Nodo %d infetta %d\n", threadIdx.x,blockIdx.x, i, neighbor);
                    }
                }
            }
            if (tid_in_warp == 0) {
                if (!is_init) {
                    hiprand_init(0, threadIdx.x, 0, &state);
                    is_init = true;
                }
                if (hiprand_uniform(&state) < q) {
                    d_Immune[i] = true; // Nodo recuperato                
                    atomicSub(d_active_infections, 1);
                    //printf("Thread %d Blocco %d: Nodo %d guarito\n", threadIdx.x, blockIdx.x, i);
                }
                else {
                    d_Levels[i] = step + 1; // Nodo può infettare anche al prossimo step
                    //printf("Thread %d Blocco %d: Nodo %d rimane infetto\n", threadIdx.x, blockIdx.x, i);
                }
            }
        }
    }
}

void simulate(double p, double q) {
    int active_infections = 1;
    int step = 0;

    //Device variables
    int* d_N, * d_L, * d_Levels;
    bool* d_Immune;
    int* d_active_infections;

    hipMalloc(&d_N, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_L, size_L * sizeof(int));
    hipMalloc(&d_Levels, num_nodes * sizeof(int));
    hipMalloc(&d_Immune, num_nodes * sizeof(bool));
    hipMalloc(&d_active_infections, sizeof(int));

    hipMemcpy(d_N, N, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, size_L * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Levels, Levels, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Immune, Immune, num_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_active_infections, &active_infections, sizeof(int), hipMemcpyHostToDevice);

    //print_status(step, active_infections, d_Levels);

    int threadsPerBlock = 64;  
    int gridSize = (num_nodes + threadsPerBlock - 1) / threadsPerBlock;

    while (active_infections > 0) {

        //Scegliendo blocchi di dimensione 32 un blocco corrisponde a un warp
        simulate_step << <gridSize, threadsPerBlock >> > (d_N, d_L, d_Levels, d_Immune, num_nodes, p, q, step, d_active_infections);
        hipDeviceSynchronize();
        hipMemcpy(&active_infections, d_active_infections, sizeof(int), hipMemcpyDeviceToHost);

        step++;
        //print_status(step, active_infections, d_Levels);
       
    }

    hipMemcpy(Levels, d_Levels, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Immune, d_Immune, num_nodes * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_N);
    hipFree(d_L);
    hipFree(d_Levels);
    hipFree(d_Immune);
    hipFree(d_active_infections);
}

int main(int argc, char* argv[]) {
    //Selezionando p=1 e q=1 otteniamo una ricerca in ampiezza
    double p = 1; // Probabilità di infezione
    double q = 1; // Probabilità di guarigione

    double start_import = cpuSecond();
    import_network(argv[1]);
    double end_import = cpuSecond();
    printf("Import time: %f seconds\n", end_import - start_import);

    //print_network();
    
	printf("Simulating with p=%f, q=%f\n", p, q);
    double start = cpuSecond();
    simulate(p, q);
    double end = cpuSecond();

	printf("Elapsed time: %f seconds\n", end - start);

    free(N);
    free(L);
    free(Levels);
    free(Immune);

    return 0;
}
